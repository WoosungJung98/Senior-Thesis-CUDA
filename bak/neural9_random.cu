#define FP float

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include <stdbool.h>
#include <pthread.h>

#define MNIST_LABEL_MAGIC 0x00000801
#define MNIST_IMAGE_MAGIC 0x00000803
#define MNIST_LABELS 10
#define BLOCK_SZ 32
#define NUM_STREAMS 4

typedef struct mnist_label_file_header_t_ {
  uint32_t magic_number;
  uint32_t num_labels;
} __attribute__((packed)) mnist_label_file_header_t;  

typedef struct mnist_image_file_header_t_ {
  uint32_t magic_number;
  uint32_t num_images;
  uint32_t num_rows;
  uint32_t num_cols;
} __attribute__((packed)) mnist_image_file_header_t;

typedef struct mnist_batch_t_ {
  FP* pixels;
  FP* labels;
  uint32_t size;
} __attribute__((packed)) mnist_batch_t;

typedef struct mnist_dataset_t_ {
  mnist_batch_t* batches;
  uint32_t num_batches;
} mnist_dataset_t;

typedef struct layer_metadata_t_ {
  FP* weights; // malloced in device (contains bias)
  FP* deltas;
  FP* outputs;
  FP* activations; // malloced in device
  uint32_t n; // number of rows in A
  uint32_t m; // number of cols in B
  uint32_t p; // number of cols in A, number of rows in B
} layer_metadata_t;

typedef struct backprop_args_t_ {
  FP* curr_derivative;
  FP* d_curr_derivative;
  FP* next_derivative;
  FP* d_next_derivative;
  FP* activation_derivative;
} backprop_args_t;

typedef struct activation_deriv_args_t_ {
  FP* d_softmax_jacobian;
  FP* d_layer_output_single;
  FP* d_softmax_derivative;
  FP* d_max_val;
  FP* d_sum;
  FP* d_relu_derivative;
} activation_deriv_args_t;

typedef struct backprop_params_t_ {
  FP* xentropy_derivative;
  int idx_in_batch;
  uint32_t batch_size;
  layer_metadata_t* layer_mtdt_arr;
  uint32_t* layer_dims_arr;
  uint32_t num_layers;
  FP learning_rate;
  hipStream_t* stream;
  backprop_args_t* args;
  activation_deriv_args_t* actv_deriv_args;
} backprop_params_t;

typedef struct cat_xentropy_args_t_ {
  FP* d_max_arr;
  FP* d_sum_arr;
  FP* xentropy_arr;
  FP* d_xentropy_arr;
} cat_xentropy_args_t;

pthread_barrier_t barrier; // barrier synchronization object
uint32_t exec_thread = 0; // determines which thread is executing
pthread_mutex_t lock = PTHREAD_MUTEX_INITIALIZER;

__device__ static float atomicMax(float* address, float val) {
  int* address_as_i = (int*) address;
  int old = *address_as_i, assumed;
  do {
    assumed = old;
    old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
  } while (assumed != old);
  return __int_as_float(old);
}

__global__ void blocked_gpu_matrixmult(FP* A, FP* B, FP* C, FP* C_activations, int n, int m, int p, bool forward_pass_flag) {
  // position within submatrix
  int C_col = threadIdx.x;
  int C_row = threadIdx.y;
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
  // A_row_abs = C_row_abs
  int A_col_abs = C_col;
  int B_row_abs = C_row;
  // B_col_abs = C_col_abs

  // total number of submatrixes
  int totalSub = (p + BLOCK_SZ - 1) / BLOCK_SZ;
  int idxSub;
  int k;
  FP Cval = 0;
 
  for(idxSub = 0; idxSub < totalSub; idxSub++) {
    // Must use fixed block size as dynamic allocation is not possible
    __shared__ FP A_shr[BLOCK_SZ][BLOCK_SZ];
    __shared__ FP B_shr[BLOCK_SZ][BLOCK_SZ];
    if(C_row_abs < n && A_col_abs < p) {
      A_shr[C_row][C_col] = A[C_row_abs * p + A_col_abs];
    }
    if(B_row_abs < p && C_col_abs < m) {
      if(forward_pass_flag) {
        if(B_row_abs == 0) {
          // Bias applied
          B_shr[C_row][C_col] = 1;
        }
        else {
          B_shr[C_row][C_col] = B[(B_row_abs - 1) * m + C_col_abs];
        }
      }
      else
        B_shr[C_row][C_col] = B[B_row_abs * m + C_col_abs];
    }
    // Synchronize
    __syncthreads();
    if(C_row_abs < n && C_col_abs < m) {
      // Multiply A submatrix and B submatrix
      if(idxSub == (totalSub - 1) && p % BLOCK_SZ != 0) {
        for(k = 0; k < p % BLOCK_SZ; k++)
          Cval += A_shr[C_row][k] * B_shr[k][C_col];
      }
      else {
        for(k = 0; k < BLOCK_SZ; k++)
          Cval += A_shr[C_row][k] * B_shr[k][C_col];
      }
    }
    // Synchronize
    __syncthreads();
    A_col_abs += BLOCK_SZ;
    B_row_abs += BLOCK_SZ;
  }
  if(C_row_abs < n && C_col_abs < m) {
    C[C_row_abs * m + C_col_abs] = Cval;
    if(forward_pass_flag) {
      // ReLU activation applied
      C_activations[C_row_abs * m + C_col_abs] = (Cval > 0) ? Cval : 0;
    }
  }
}

FP randn(double mu, double sigma) {
  double U1, U2, W, mult;
  static double X1, X2;
  static int call = 0;
 
  if (call == 1) {
    call = !call;
    return (mu + sigma * (double) X2);
  }
 
  do {
    U1 = -1 + ((double) rand () / RAND_MAX) * 2;
    U2 = -1 + ((double) rand () / RAND_MAX) * 2;
    W = pow (U1, 2) + pow (U2, 2);
  }
  while (W >= 1 || W == 0);
 
  mult = sqrt ((-2 * log (W)) / W);
  X1 = U1 * mult;
  X2 = U2 * mult;
 
  call = !call;
 
  return (FP)(mu + sigma * (double) X1);
}

uint8_t* read_binary_file(const char* filename) {
  FILE *fileptr;
  uint8_t *buffer;
  long filelen;

  fileptr = fopen(filename, "rb");
  fseek(fileptr, 0, SEEK_END);
  filelen = ftell(fileptr);
  rewind(fileptr);

  buffer = (uint8_t *)malloc(filelen * sizeof(uint8_t));
  fread(buffer, filelen, 1, fileptr);
  fclose(fileptr);
  return buffer;
}

uint32_t big_to_little_endian(uint32_t num) {
  return (
    ((num & 0xFF000000) >> 24) |
    ((num & 0x00FF0000) >>  8) |
    ((num & 0x0000FF00) <<  8) |
    ((num & 0x000000FF) << 24)
  );
}

void images_header_to_little_endian(mnist_image_file_header_t* images_header) {
  images_header->magic_number = big_to_little_endian(images_header->magic_number);
  images_header->num_images = big_to_little_endian(images_header->num_images);
  images_header->num_rows = big_to_little_endian(images_header->num_rows);
  images_header->num_cols = big_to_little_endian(images_header->num_cols);
}

void labels_header_to_little_endian(mnist_label_file_header_t* labels_header) {
  labels_header->magic_number = big_to_little_endian(labels_header->magic_number);
  labels_header->num_labels = big_to_little_endian(labels_header->num_labels);

}

void init_dataset(mnist_dataset_t* dataset, mnist_image_file_header_t* images_header, uint8_t* images_raw, mnist_label_file_header_t* labels_header, uint8_t* labels_raw, uint32_t batch_size) {
  if(images_header->num_images % batch_size == 0)
    dataset->num_batches = images_header->num_images / batch_size;
  else
    dataset->num_batches = images_header->num_images / batch_size + 1;
  dataset->batches = (mnist_batch_t*)malloc(dataset->num_batches * sizeof(mnist_batch_t));
  int image_total_pixels = images_header->num_rows * images_header->num_cols;
  int image_dataset_total_bytes = sizeof(mnist_image_file_header_t) + image_total_pixels * images_header->num_images;
  int actual_batch_size;
  int batch_idx = 0;
  int pixel_idx, label_idx;
  int raw_label_idx = sizeof(mnist_label_file_header_t);
  int i, j, k;

  for(i=sizeof(mnist_image_file_header_t); i<image_dataset_total_bytes; i+=image_total_pixels * batch_size) {
    if(i + image_total_pixels * batch_size > image_dataset_total_bytes)
      actual_batch_size = (image_dataset_total_bytes - i) / image_total_pixels;
    else
      actual_batch_size = batch_size;
    dataset->batches[batch_idx].pixels = (FP*)malloc(actual_batch_size * image_total_pixels * sizeof(FP));
    dataset->batches[batch_idx].labels = (FP*)malloc(actual_batch_size * MNIST_LABELS * sizeof(FP));
    dataset->batches[batch_idx].size = actual_batch_size;
    pixel_idx = 0;
    for(j=i; j<i + image_total_pixels; j++) {
      for(k=j; k<i + image_total_pixels * actual_batch_size; k+=image_total_pixels) {
        dataset->batches[batch_idx].pixels[pixel_idx] = images_raw[k] / (FP)255;
        pixel_idx++;
      }
    }
    label_idx = 0;
    for(k=0; k<MNIST_LABELS; k++) {
      for(j=raw_label_idx; j<raw_label_idx + actual_batch_size; j++) {
        dataset->batches[batch_idx].labels[label_idx] = labels_raw[j] == k;
        label_idx++;
      }
    }
    raw_label_idx += actual_batch_size; 
    batch_idx++;
  }
}

void free_dataset(mnist_dataset_t* dataset) {
  int batch_idx;
  for(batch_idx=0; batch_idx<dataset->num_batches; batch_idx++) {
    free(dataset->batches[batch_idx].pixels);
    free(dataset->batches[batch_idx].labels);
  }
  free(dataset->batches);
}

layer_metadata_t* init_layer_metadata_arr(uint32_t* layer_dims_arr, uint32_t num_layers, uint32_t batch_size) {
  layer_metadata_t* layer_mtdt_arr = (layer_metadata_t*)malloc(num_layers * sizeof(layer_metadata_t));
  uint32_t prev_dim, curr_dim;
  int i, k;

  layer_mtdt_arr[0].weights = NULL;
  layer_mtdt_arr[0].deltas = NULL;
  layer_mtdt_arr[0].outputs = NULL;
  hipMalloc((void**)&layer_mtdt_arr[0].activations, layer_dims_arr[0] * batch_size * sizeof(FP));

  for(i=1; i<num_layers; i++) {
    prev_dim = layer_dims_arr[i-1];
    curr_dim = layer_dims_arr[i];
    hipMalloc((void**)&layer_mtdt_arr[i].weights, (prev_dim + 1) * curr_dim * sizeof(FP));
    FP* init_weights = (FP*)malloc((prev_dim + 1) * curr_dim * sizeof(FP));
    for(k=0; k<(prev_dim + 1) * curr_dim; k++) {
      if(k % (prev_dim + 1) == 0) {
        init_weights[k] = 0;
        continue;
      }
      init_weights[k] = randn(0, sqrt(2/(double) prev_dim));
    }
    hipMemcpy(layer_mtdt_arr[i].weights, init_weights, (prev_dim + 1) * curr_dim * sizeof(FP), hipMemcpyHostToDevice);
    free(init_weights);
    hipMalloc((void**)&layer_mtdt_arr[i].deltas, (prev_dim + 1) * curr_dim * sizeof(FP));
    FP* init_deltas = (FP*)malloc((prev_dim + 1) * curr_dim * sizeof(FP));
    for(k=0; k<(prev_dim + 1) * curr_dim; k++) {
      init_deltas[k] = 0;
    }
    hipMemcpy(layer_mtdt_arr[i].deltas, init_deltas, (prev_dim + 1) * curr_dim * sizeof(FP), hipMemcpyHostToDevice);
    free(init_deltas);
    hipMalloc((void**)&layer_mtdt_arr[i].outputs, curr_dim * batch_size * sizeof(FP));
    hipMalloc((void**)&layer_mtdt_arr[i].activations, curr_dim * batch_size * sizeof(FP));
    layer_mtdt_arr[i].n = curr_dim;
    layer_mtdt_arr[i].m = batch_size;
    layer_mtdt_arr[i].p = prev_dim + 1;
  }
  
  return layer_mtdt_arr;
}

void free_layer_metadata_arr(layer_metadata_t* layer_mtdt_arr, uint32_t num_layers) {
  int i;
  hipFree(layer_mtdt_arr[0].activations);
  for(i=1; i<num_layers; i++) {
    hipFree(layer_mtdt_arr[i].weights);
    hipFree(layer_mtdt_arr[i].deltas);
    hipFree(layer_mtdt_arr[i].outputs);
    hipFree(layer_mtdt_arr[i].activations);
  }
  free(layer_mtdt_arr);
}

__global__ void calc_max_output_layer(FP* C, int n, int m, FP* d_max_arr) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
 
  if(C_row_abs < n && C_col_abs < m)
    atomicMax(&d_max_arr[C_col_abs], C[C_row_abs * m + C_col_abs]);
}

__global__ void calc_sum_output_layer(FP* C, int n, int m, FP* d_max_arr, FP* d_sum_arr) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
 
  if(C_row_abs < n && C_col_abs < m)
    atomicAdd(&d_sum_arr[C_col_abs], ::expf(C[C_row_abs * m + C_col_abs] - d_max_arr[C_col_abs])); 
}

__global__ void softmax_output_layer(FP* C, FP* C_activations, int n, int m, FP* d_max_arr, FP* d_sum_arr, FP* d_xentropy_arr) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
 
  if(C_row_abs < n && C_col_abs < m) {
    C_activations[C_row_abs * m + C_col_abs] = ::expf(C[C_row_abs * m + C_col_abs] - d_max_arr[C_col_abs] - ::logf(d_sum_arr[C_col_abs]));
    d_xentropy_arr[C_col_abs] = 0;
  }
}

__global__ void crossentropy_output_layer(FP* C_activations, int n, int m, FP* d_labels, FP* d_max_arr, FP* d_sum_arr, FP* d_xentropy_arr) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
 
  if(C_row_abs < n && C_col_abs < m) {
    d_max_arr[C_col_abs] = -INFINITY;
    d_sum_arr[C_col_abs] = 0;
    atomicAdd(&d_xentropy_arr[C_col_abs], -1 * d_labels[C_row_abs * m + C_col_abs] * ::logf(C_activations[C_row_abs * m + C_col_abs]));
  }
}

FP* calc_categorical_xentropy(FP* output_layer_outputs, FP* output_layer_activations, uint32_t batch_size, FP* d_labels, cat_xentropy_args_t* args) {
  FP* d_max_arr = args->d_max_arr;
  FP* d_sum_arr = args->d_sum_arr;
  FP* xentropy_arr = args->xentropy_arr;
  FP* d_xentropy_arr = args->d_xentropy_arr;
  
  dim3 Grid;
  Grid.x = (batch_size + BLOCK_SZ - 1) / BLOCK_SZ;
  Grid.y = (MNIST_LABELS + BLOCK_SZ - 1) / BLOCK_SZ; 
  dim3 Block(BLOCK_SZ, BLOCK_SZ);

  calc_max_output_layer<<<Grid,Block>>>(output_layer_outputs, MNIST_LABELS, batch_size, d_max_arr);
  calc_sum_output_layer<<<Grid,Block>>>(output_layer_outputs, MNIST_LABELS, batch_size, d_max_arr, d_sum_arr);
  softmax_output_layer<<<Grid,Block>>>(output_layer_outputs, output_layer_activations, MNIST_LABELS, batch_size, d_max_arr, d_sum_arr, d_xentropy_arr);
  
  crossentropy_output_layer<<<Grid,Block>>>(output_layer_activations, MNIST_LABELS, batch_size, d_labels, d_max_arr, d_sum_arr, d_xentropy_arr);

  hipMemcpy(xentropy_arr, d_xentropy_arr, batch_size * sizeof(FP), hipMemcpyDeviceToHost);

  return xentropy_arr;
}

__global__ void extract_single_output(FP* C, int n, int m, FP* C_full, int batch_size, int idx_in_batch, FP* d_max_val, FP* d_sum) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
 
  if(C_row_abs < n && C_col_abs < m) {
    if(C_row_abs == 0) {
      C[C_row_abs] = 1;
      if(d_max_val != NULL) *d_max_val = -INFINITY;
      if(d_sum != NULL) *d_sum = 0;
    }
    else
      C[C_row_abs] = C_full[(C_row_abs - 1) * batch_size + idx_in_batch];
  }
}

__global__ void calc_single_output_max(FP* C, int n, int m, FP* d_max_val) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
 
  if(C_row_abs < n && C_col_abs < m)
    atomicMax(d_max_val, C[C_row_abs * m + C_col_abs]);
}

__global__ void calc_single_output_sum(FP* C, int n, int m, FP* d_max_val, FP* d_sum) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
 
  if(C_row_abs < n && C_col_abs < m)
    atomicAdd(d_sum, ::expf(C[C_row_abs] - *d_max_val));
}

__global__ void calc_softmax_jacobian(FP* C, int n, int m, FP* d_layer_output_single, FP* d_max_val, FP* d_sum) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
 
  if(C_row_abs < n && C_col_abs < m) {
    FP softmax_output_i = ::expf(d_layer_output_single[C_row_abs] - *d_max_val - ::logf(*d_sum));
    if(C_row_abs == C_col_abs)
      C[C_row_abs * m + C_col_abs] = softmax_output_i * (1 - softmax_output_i);
    else {
      FP softmax_output_j = ::expf(d_layer_output_single[C_col_abs] - *d_max_val - ::logf(*d_sum));
      C[C_row_abs * m + C_col_abs] = -1 * softmax_output_i * softmax_output_j;
    }
  }
}

FP* calc_softmax_derivative(FP* d_layer_outputs, int idx_in_batch, uint32_t batch_size, uint32_t layer_dim, hipStream_t* stream, activation_deriv_args_t* args) {
  FP* d_softmax_jacobian = args->d_softmax_jacobian;
  FP* d_layer_output_single = args->d_layer_output_single;
  FP* d_softmax_derivative = args->d_softmax_derivative;
  FP* d_max_val = args->d_max_val;
  FP* d_sum = args->d_sum;

  uint32_t curr_thread = idx_in_batch % NUM_STREAMS;

  dim3 Grid;
  dim3 Block(BLOCK_SZ, BLOCK_SZ);

  Grid.x = (1 + BLOCK_SZ - 1) / BLOCK_SZ;
  Grid.y = (layer_dim + BLOCK_SZ - 1) / BLOCK_SZ;
  extract_single_output<<<Grid,Block,0,*stream>>>(d_layer_output_single, layer_dim, 1, d_layer_outputs, batch_size, idx_in_batch, d_max_val, d_sum);

  calc_single_output_max<<<Grid,Block,0,*stream>>>(d_layer_output_single, layer_dim, 1, d_max_val);

  calc_single_output_sum<<<Grid,Block,0,*stream>>>(d_layer_output_single, layer_dim, 1, d_max_val, d_sum);

  Grid.x = (layer_dim + BLOCK_SZ - 1) / BLOCK_SZ;
  Grid.y = (layer_dim + BLOCK_SZ - 1) / BLOCK_SZ;
  calc_softmax_jacobian<<<Grid,Block,0,*stream>>>(d_softmax_jacobian, layer_dim, layer_dim, d_layer_output_single, d_max_val, d_sum);
  
  Grid.x = (layer_dim + BLOCK_SZ - 1) / BLOCK_SZ;
  Grid.y = (1 + BLOCK_SZ - 1) / BLOCK_SZ;
  blocked_gpu_matrixmult<<<Grid,Block,0,*stream>>>(d_layer_output_single, d_softmax_jacobian, d_softmax_derivative, NULL, 1, layer_dim, layer_dim, false);
  
  return d_softmax_derivative;
}

__global__ void calc_relu_derivative_gpu(FP* C, int n, int m, FP* d_layer_output_single) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
 
  if(C_row_abs < n && C_col_abs < m)
    C[C_row_abs] = (d_layer_output_single[C_row_abs] > 0) ? 1 : 0;
}

FP* calc_relu_derivative(FP* d_layer_outputs, int idx_in_batch, uint32_t batch_size, uint32_t layer_dim, hipStream_t* stream, activation_deriv_args_t* args) {
  FP* d_layer_output_single = args->d_layer_output_single;
  FP* d_relu_derivative = args->d_relu_derivative;

  uint32_t curr_thread = idx_in_batch % NUM_STREAMS;

  dim3 Grid;
  dim3 Block(BLOCK_SZ, BLOCK_SZ);

  Grid.x = (1 + BLOCK_SZ - 1) / BLOCK_SZ;
  Grid.y = (layer_dim + BLOCK_SZ - 1) / BLOCK_SZ;
  extract_single_output<<<Grid,Block,0,*stream>>>(d_layer_output_single, layer_dim, 1, d_layer_outputs, batch_size, idx_in_batch, NULL, NULL);

  calc_relu_derivative_gpu<<<Grid,Block,0,*stream>>>(d_relu_derivative, layer_dim, 1, d_layer_output_single);

  return d_relu_derivative;
}

__global__ void update_deltas_gpu(FP* curr_derivative, FP* activations, int idx_in_batch, int batch_size, FP* deltas, int n, int m, FP learning_rate) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
  int actv_idx = (C_col_abs - 1) * batch_size + idx_in_batch;
  FP actv_value = (C_col_abs == 0) ? 1 : activations[actv_idx]; 
 
  if(C_row_abs < n && C_col_abs < m)
    atomicAdd(&deltas[C_row_abs * m + C_col_abs], learning_rate * curr_derivative[C_row_abs] * actv_value);
}

void update_deltas(FP* d_curr_derivative, FP* d_activations, int idx_in_batch, int batch_size, FP* d_deltas, int n, int m, FP learning_rate, hipStream_t* stream) {
  dim3 Block(BLOCK_SZ, BLOCK_SZ);
  dim3 Grid;
  Grid.x = (m + BLOCK_SZ - 1) / BLOCK_SZ;
  Grid.y = (n + BLOCK_SZ - 1) / BLOCK_SZ;
  update_deltas_gpu<<<Grid,Block,0,*stream>>>(d_curr_derivative, d_activations, idx_in_batch, batch_size, d_deltas, n, m, learning_rate);
}

void* stochastic_backprop(void* backprop_params) {
  backprop_params_t* params = (backprop_params_t*)backprop_params;
  FP* xentropy_derivative = params->xentropy_derivative;
  int idx_in_batch = params->idx_in_batch;
  uint32_t batch_size = params->batch_size;
  layer_metadata_t* layer_mtdt_arr = params->layer_mtdt_arr;
  uint32_t* layer_dims_arr = params->layer_dims_arr;
  uint32_t num_layers = params->num_layers;
  FP learning_rate = params->learning_rate;
  hipStream_t* stream = params->stream;
  backprop_args_t* args = params->args;
  activation_deriv_args_t* actv_deriv_args = params->actv_deriv_args;
  
  FP* curr_derivative = args->curr_derivative;
  FP* d_curr_derivative = args->d_curr_derivative;
  FP* next_derivative = args->next_derivative;
  FP* d_next_derivative = args->d_next_derivative;
  FP* activation_derivative = args->activation_derivative;

  uint32_t curr_thread = idx_in_batch % NUM_STREAMS;

  // Extract the cross entropy derivative for a specific observation in batch based on idx_in_batch
  int i, j;
  j = 0;
  for(i=idx_in_batch; i<MNIST_LABELS * batch_size; i+=batch_size, j++) {
    curr_derivative[j] = xentropy_derivative[i];
  }

  hipMemcpyAsync(d_curr_derivative, curr_derivative, MNIST_LABELS * sizeof(FP), hipMemcpyHostToDevice, *stream);

  uint32_t curr_dim, next_dim;
  FP* d_activation_derivative;
  dim3 Block(BLOCK_SZ, BLOCK_SZ);
  dim3 Grid;

  for(i=num_layers - 1; i>=2; i--) {
    curr_dim = layer_dims_arr[i];
    next_dim = layer_dims_arr[i - 1] + 1;  
       
    Grid.x = (next_dim + BLOCK_SZ - 1) / BLOCK_SZ;
    Grid.y = (1 + BLOCK_SZ - 1) / BLOCK_SZ;

    if(i == num_layers - 1)
      d_activation_derivative = calc_softmax_derivative(layer_mtdt_arr[i - 1].outputs, idx_in_batch, batch_size, next_dim, stream, actv_deriv_args);
    else
      d_activation_derivative = calc_relu_derivative(layer_mtdt_arr[i - 1].outputs, idx_in_batch, batch_size, next_dim, stream, actv_deriv_args);
    
    blocked_gpu_matrixmult<<<Grid,Block,0,*stream>>>(d_curr_derivative, layer_mtdt_arr[i].weights, d_next_derivative, NULL, 1, next_dim, curr_dim, false);
    
    update_deltas(d_curr_derivative, layer_mtdt_arr[i - 1].activations, idx_in_batch, batch_size, layer_mtdt_arr[i].deltas, curr_dim, next_dim, learning_rate, stream);

    hipMemcpyAsync(activation_derivative, d_activation_derivative, next_dim * sizeof(FP), hipMemcpyDeviceToHost, *stream);
    
    hipMemcpyAsync(next_derivative, d_next_derivative, next_dim * sizeof(FP), hipMemcpyDeviceToHost, *stream);

    for(j=0; j<next_dim; j++) {
      next_derivative[j] *= activation_derivative[j];
    }
    for(j=1; j<next_dim; j++) {
      curr_derivative[j-1] = next_derivative[j];
    }

    hipMemcpyAsync(d_curr_derivative, curr_derivative, (next_dim - 1) * sizeof(FP), hipMemcpyHostToDevice, *stream);
  }

  update_deltas(d_curr_derivative, layer_mtdt_arr[0].activations, idx_in_batch, batch_size, layer_mtdt_arr[1].deltas, layer_dims_arr[1], layer_dims_arr[0] + 1, learning_rate, stream);
  
  return NULL;
}

__global__ void update_weights_biases(FP* weights, FP* deltas, FP batch_size, int n, int m) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
 
  if(C_row_abs < n && C_col_abs < m) {
    weights[C_row_abs * m + C_col_abs] -= ::fdividef(deltas[C_row_abs * m + C_col_abs], batch_size);
    deltas[C_row_abs * m + C_col_abs] = 0;
  }
}

FP calc_accuracy(FP* activations, FP* labels, uint32_t batch_size) {
  int num_correct = 0;
  int i, j;
  int actv_idx;
  FP max_val;
  int max_i;
  for(j=0; j<batch_size; j++) {
    actv_idx = j;
    max_val = -INFINITY;
    for(i=0; i<MNIST_LABELS; i++) {
      if(activations[actv_idx] > max_val) {
        max_val = activations[actv_idx];
        max_i = i;
      }
      actv_idx += batch_size;
    }
    if((int)round(labels[max_i * batch_size + j]) == 1)
      num_correct++;
  }
  return num_correct / (FP)batch_size;
}

__global__ void calc_xentropy_derivative(FP* C, int n, int m, FP* d_labels, FP* derivative) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
 
  if(C_row_abs < n && C_col_abs < m)
    derivative[C_row_abs * m + C_col_abs] = C[C_row_abs * m + C_col_abs] - d_labels[C_row_abs * m + C_col_abs];
}

void train_mnist(mnist_dataset_t* train_dataset, layer_metadata_t* layer_mtdt_arr, uint32_t* layer_dims_arr, uint32_t num_layers, FP learning_rate) {
  dim3 Grid;
  dim3 Block(BLOCK_SZ, BLOCK_SZ);
  int i, j, k;
  uint32_t actual_batch_size;
  FP* xentropy_arr;
  FP* output_layer_activations = (FP*)malloc(MNIST_LABELS * train_dataset->batches[0].size * sizeof(FP));
  FP batch_train_accuracy;
  backprop_params_t* backprop_params_arr = (backprop_params_t*)malloc(sizeof(backprop_params_t) * NUM_STREAMS);
  pthread_t threads[NUM_STREAMS];
  hipStream_t streams[NUM_STREAMS];
  for(i=0; i<NUM_STREAMS; i++) {
    hipStreamCreate(&streams[i]);
  }
  FP* d_labels;
  hipMalloc((void**)&d_labels, MNIST_LABELS * train_dataset->batches[0].size * sizeof(FP));
  FP* d_xentropy_derivative;
  hipMalloc((void**)&d_xentropy_derivative, MNIST_LABELS * train_dataset->batches[0].size * sizeof(FP));
  FP* xentropy_derivative;
  hipHostMalloc((void**)&xentropy_derivative, MNIST_LABELS * train_dataset->batches[0].size * sizeof(FP), hipHostMallocDefault);

  uint32_t max_dim = 0;
  for(i=num_layers - 1; i>=2; i--) {
    if(layer_dims_arr[i - 1] + 1 > max_dim)
      max_dim = layer_dims_arr[i - 1] + 1;
  }
  backprop_args_t* backprop_args_arr = (backprop_args_t*)malloc(NUM_STREAMS * sizeof(backprop_args_t));
  for(i=0; i<NUM_STREAMS; i++) {
    hipHostMalloc((void**)&backprop_args_arr[i].curr_derivative, (max_dim - 1) * sizeof(FP), hipHostMallocDefault);
    hipMalloc((void**)&backprop_args_arr[i].d_curr_derivative, (max_dim - 1) * sizeof(FP));
    hipHostMalloc((void**)&backprop_args_arr[i].next_derivative, max_dim * sizeof(FP), hipHostMallocDefault);
    hipMalloc((void**)&backprop_args_arr[i].d_next_derivative, max_dim * sizeof(FP));
    hipHostMalloc((void**)&backprop_args_arr[i].activation_derivative, max_dim * sizeof(FP), hipHostMallocDefault);
  }

  activation_deriv_args_t* actv_deriv_args_arr = (activation_deriv_args_t*)malloc(NUM_STREAMS * sizeof(activation_deriv_args_t));
  for(i=0; i<NUM_STREAMS; i++) {
    hipMalloc((void**)&actv_deriv_args_arr[i].d_softmax_jacobian, max_dim * max_dim * sizeof(FP));
    hipMalloc((void**)&actv_deriv_args_arr[i].d_layer_output_single, max_dim * sizeof(FP));
    hipMalloc((void**)&actv_deriv_args_arr[i].d_softmax_derivative, max_dim * sizeof(FP));
    hipMalloc((void**)&actv_deriv_args_arr[i].d_max_val, sizeof(FP));
    hipMalloc((void**)&actv_deriv_args_arr[i].d_sum, sizeof(FP));
    hipMalloc((void**)&actv_deriv_args_arr[i].d_relu_derivative, max_dim * sizeof(FP));
  }

  cat_xentropy_args_t xentropy_args;
  hipMalloc((void**)&xentropy_args.d_max_arr, train_dataset->batches[0].size * sizeof(FP));
  hipMalloc((void**)&xentropy_args.d_sum_arr, train_dataset->batches[0].size * sizeof(FP));
  hipHostMalloc((void**)&xentropy_args.xentropy_arr, train_dataset->batches[0].size * sizeof(FP), hipHostMallocDefault);
  hipMalloc((void**)&xentropy_args.d_xentropy_arr, train_dataset->batches[0].size * sizeof(FP));
  FP* max_arr = (FP*)malloc(train_dataset->batches[0].size * sizeof(FP));
  FP* sum_arr = (FP*)malloc(train_dataset->batches[0].size * sizeof(FP));
  for(i=0; i<train_dataset->batches[0].size; i++) {
    max_arr[i] = -INFINITY;
    sum_arr[i] = 0;
  }
  hipMemcpy(xentropy_args.d_max_arr, max_arr, train_dataset->batches[0].size * sizeof(FP), hipMemcpyHostToDevice);
  hipMemcpy(xentropy_args.d_sum_arr, sum_arr, train_dataset->batches[0].size * sizeof(FP), hipMemcpyHostToDevice);
  free(max_arr);
  free(sum_arr);

  if(pthread_barrier_init(&barrier, NULL, NUM_STREAMS) != 0) {
    printf("\n barrier init has failed\n");
    return;
  }
  
  // Mini Batch Gradient Descent
  for(i=0; i<train_dataset->num_batches; i++) {
    actual_batch_size = train_dataset->batches[i].size; 
    hipMemcpy(layer_mtdt_arr[0].activations, train_dataset->batches[i].pixels, layer_dims_arr[0] * actual_batch_size * sizeof(FP), hipMemcpyHostToDevice);
    for(j=1; j<num_layers; j++) {
      layer_mtdt_arr[j].m = actual_batch_size;
      Grid.x = (layer_mtdt_arr[j].m + BLOCK_SZ - 1) / BLOCK_SZ;
      Grid.y = (layer_mtdt_arr[j].n + BLOCK_SZ - 1) / BLOCK_SZ;
      // Calculate outputs and ReLU activations for next layer
      blocked_gpu_matrixmult<<<Grid,Block>>>(layer_mtdt_arr[j].weights, layer_mtdt_arr[j-1].activations, layer_mtdt_arr[j].outputs, layer_mtdt_arr[j].activations, layer_mtdt_arr[j].n, layer_mtdt_arr[j].m, layer_mtdt_arr[j].p, true);
    }
    
    // Apply softmax to last layer and calculate categorical cross entropy loss
    hipMemcpy(d_labels, train_dataset->batches[i].labels, MNIST_LABELS * actual_batch_size * sizeof(FP), hipMemcpyHostToDevice);
    xentropy_arr = calc_categorical_xentropy(layer_mtdt_arr[num_layers - 1].outputs, layer_mtdt_arr[num_layers - 1].activations, actual_batch_size, d_labels, &xentropy_args);
    
    // Calculate training accuracy for batch
    hipMemcpy(output_layer_activations, layer_mtdt_arr[num_layers - 1].activations, MNIST_LABELS * actual_batch_size * sizeof(FP), hipMemcpyDeviceToHost);
    batch_train_accuracy = calc_accuracy(output_layer_activations, train_dataset->batches[i].labels, actual_batch_size);
    printf("Batch %d Train Accuracy: %.2f\n", i + 1, batch_train_accuracy);
    
    // Stochastic backpropagation for each example in batch
    Grid.x = (actual_batch_size + BLOCK_SZ - 1) / BLOCK_SZ;
    Grid.y = (MNIST_LABELS + BLOCK_SZ - 1) / BLOCK_SZ;
    calc_xentropy_derivative<<<Grid,Block>>>(layer_mtdt_arr[num_layers - 1].activations, MNIST_LABELS, actual_batch_size, d_labels, d_xentropy_derivative);
    hipMemcpy(xentropy_derivative, d_xentropy_derivative, MNIST_LABELS * actual_batch_size * sizeof(FP), hipMemcpyDeviceToHost);

    for(j=0; j<actual_batch_size; j+=NUM_STREAMS) {
      for(k=0; k<NUM_STREAMS; k++) {
        if(j + k == actual_batch_size) break;
        backprop_params_arr[k].xentropy_derivative = xentropy_derivative;
        backprop_params_arr[k].idx_in_batch = j + k;
        backprop_params_arr[k].batch_size = actual_batch_size;
        backprop_params_arr[k].layer_mtdt_arr = layer_mtdt_arr;
        backprop_params_arr[k].layer_dims_arr = layer_dims_arr;
        backprop_params_arr[k].num_layers = num_layers;
        backprop_params_arr[k].learning_rate = learning_rate;
        backprop_params_arr[k].stream = &streams[k];
        backprop_params_arr[k].args = &backprop_args_arr[k];
        backprop_params_arr[k].actv_deriv_args = &actv_deriv_args_arr[k];
        if(pthread_create(&threads[k], NULL, stochastic_backprop, (void*)&backprop_params_arr[k])) {
          printf("Error creating threadn\n");
          return;
        }
      }
      for(k=0; k<NUM_STREAMS; k++) {
        if(j + k == actual_batch_size) break;
        if(pthread_join(threads[k], NULL)) {
          printf("Error joining threadn\n");
          return;
        }
      }
    }

    // Update weights and biases with computed average deltas
    for(j=1; j<num_layers; j++) {
      Grid.x = ((layer_dims_arr[j - 1] + 1) + BLOCK_SZ - 1) / BLOCK_SZ;
      Grid.y = (layer_dims_arr[j] + BLOCK_SZ - 1) / BLOCK_SZ;
      update_weights_biases<<<Grid,Block>>>(layer_mtdt_arr[j].weights, layer_mtdt_arr[j].deltas, (FP)actual_batch_size, layer_dims_arr[j], layer_dims_arr[j - 1] + 1);
    }
    // PRINT START
    /*printf("\n\nBatch %d Cross Entropy\n\n", i);
    for(j=0; j<actual_batch_size; j++) {
      printf("%.2f ", xentropy_arr[j]);
    }
    printf("\n");*/
    // PRINT END
  }
  
  for(i=0; i<NUM_STREAMS; i++) {
    hipStreamDestroy(streams[i]);
  }
  free(output_layer_activations);
  free(backprop_params_arr);
  hipFree(d_labels);
  hipFree(d_xentropy_derivative);
  hipHostFree(xentropy_derivative);

  for(i=0; i<NUM_STREAMS; i++) {
    hipHostFree(backprop_args_arr[i].curr_derivative);
    hipFree(backprop_args_arr[i].d_curr_derivative);
    hipHostFree(backprop_args_arr[i].next_derivative);
    hipFree(backprop_args_arr[i].d_next_derivative);
    hipHostFree(backprop_args_arr[i].activation_derivative);
  }
  free(backprop_args_arr);

  for(i=0; i<NUM_STREAMS; i++) {
    hipFree(actv_deriv_args_arr[i].d_softmax_jacobian);
    hipFree(actv_deriv_args_arr[i].d_layer_output_single);
    hipFree(actv_deriv_args_arr[i].d_softmax_derivative);
    hipFree(actv_deriv_args_arr[i].d_max_val);
    hipFree(actv_deriv_args_arr[i].d_sum);
    hipFree(actv_deriv_args_arr[i].d_relu_derivative);
  }
  free(actv_deriv_args_arr);

  hipFree(xentropy_args.d_max_arr);
  hipFree(xentropy_args.d_sum_arr);
  hipHostFree(xentropy_args.xentropy_arr);
  hipFree(xentropy_args.d_xentropy_arr);

  pthread_barrier_destroy(&barrier);
}

void test_mnist(mnist_dataset_t* test_dataset, layer_metadata_t* layer_mtdt_arr, uint32_t* layer_dims_arr, uint32_t num_layers) {
  dim3 Grid;
  dim3 Block(BLOCK_SZ, BLOCK_SZ); 
  int i, j;
  uint32_t actual_batch_size;
  FP* xentropy_arr;
  FP* output_layer_activations = (FP*)malloc(MNIST_LABELS * test_dataset->batches[0].size * sizeof(FP));
  FP batch_test_accuracy;
  FP overall_test_accuracy = 0;
  FP* d_labels;
  hipMalloc((void**)&d_labels, MNIST_LABELS * test_dataset->batches[0].size * sizeof(FP));

  cat_xentropy_args_t xentropy_args;
  hipMalloc((void**)&xentropy_args.d_max_arr, test_dataset->batches[0].size * sizeof(FP));
  hipMalloc((void**)&xentropy_args.d_sum_arr, test_dataset->batches[0].size * sizeof(FP));
  hipHostMalloc((void**)&xentropy_args.xentropy_arr, test_dataset->batches[0].size * sizeof(FP), hipHostMallocDefault);
  hipMalloc((void**)&xentropy_args.d_xentropy_arr, test_dataset->batches[0].size * sizeof(FP));
  FP* max_arr = (FP*)malloc(test_dataset->batches[0].size * sizeof(FP));
  FP* sum_arr = (FP*)malloc(test_dataset->batches[0].size * sizeof(FP));
  for(i=0; i<test_dataset->batches[0].size; i++) {
    max_arr[i] = -INFINITY;
    sum_arr[i] = 0;
  }
  hipMemcpy(xentropy_args.d_max_arr, max_arr, test_dataset->batches[0].size * sizeof(FP), hipMemcpyHostToDevice);
  hipMemcpy(xentropy_args.d_sum_arr, sum_arr, test_dataset->batches[0].size * sizeof(FP), hipMemcpyHostToDevice);
  free(max_arr);
  free(sum_arr);
 
  for(i=0; i<test_dataset->num_batches; i++) {
    actual_batch_size = test_dataset->batches[i].size; 
    hipMemcpy(layer_mtdt_arr[0].activations, test_dataset->batches[i].pixels, layer_dims_arr[0] * actual_batch_size * sizeof(FP), hipMemcpyHostToDevice);
    for(j=1; j<num_layers; j++) {
      layer_mtdt_arr[j].m = actual_batch_size;
      Grid.x = (layer_mtdt_arr[j].m + BLOCK_SZ - 1) / BLOCK_SZ;
      Grid.y = (layer_mtdt_arr[j].n + BLOCK_SZ - 1) / BLOCK_SZ;
      // Calculate outputs and ReLU activations for next layer
      blocked_gpu_matrixmult<<<Grid,Block>>>(layer_mtdt_arr[j].weights, layer_mtdt_arr[j-1].activations, layer_mtdt_arr[j].outputs, layer_mtdt_arr[j].activations, layer_mtdt_arr[j].n, layer_mtdt_arr[j].m, layer_mtdt_arr[j].p, true);
    }

    // Apply softmax to the last layer and calculate categorical cross entropy loss
    hipMemcpy(d_labels, test_dataset->batches[i].labels, MNIST_LABELS * actual_batch_size * sizeof(FP), hipMemcpyHostToDevice);
    xentropy_arr = calc_categorical_xentropy(layer_mtdt_arr[num_layers - 1].outputs, layer_mtdt_arr[num_layers - 1].activations, actual_batch_size, d_labels, &xentropy_args);
    
    // Calculate test accuracy for batch
    hipMemcpy(output_layer_activations, layer_mtdt_arr[num_layers - 1].activations, MNIST_LABELS * actual_batch_size * sizeof(FP), hipMemcpyDeviceToHost);
    batch_test_accuracy = calc_accuracy(output_layer_activations, test_dataset->batches[i].labels, actual_batch_size);
    printf("Batch %d Test Accuracy: %.2f\n", i + 1, batch_test_accuracy);
    overall_test_accuracy += batch_test_accuracy;
    
    // PRINT START
    /*printf("\n\nBatch %d Cross Entropy\n\n", i);
    for(j=0; j<actual_batch_size; j++) {
      printf("%.2f ", xentropy_arr[j]);
    }
    printf("\n");*/
    // PRINT END
  }

  overall_test_accuracy /= (FP)test_dataset->num_batches;
  printf("Overall Test Accuracy: %.2f\n", overall_test_accuracy);
  free(output_layer_activations);
  hipFree(d_labels);
  hipFree(xentropy_args.d_max_arr);
  hipFree(xentropy_args.d_sum_arr);
  hipHostFree(xentropy_args.xentropy_arr);
  hipFree(xentropy_args.d_xentropy_arr);
}

int main(int argc, char *argv[]) {
  int i; // loop counters

  int gpucount = 0; // Count of available GPUs
  int gpunum = 0; // Device number to use 
  
  uint32_t* layer_dims_arr;

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else {
     printf("Device count = %d\n",gpucount);
  }
   
  if (argc < 5) {
    printf("Usage: neural <epochs> <learning rate> <batch size> <layer dim 1> <layer dim 2> ... \n");
    exit (-1);
  }
  if (argc < 6) {
    printf("Must specify input and output layer dims\n");
    exit(-1);
  }
  if (argc < 7) {
    printf("Must specify at least one hidden layer\n");
    exit(-1);
  }

  if(BLOCK_SZ * BLOCK_SZ > 1024) {
    printf("Error, too many threads in block\n");
    exit(-1);
  }
  
  int epochs = atoi(argv[1]);
  FP learning_rate = (FP)atof(argv[2]);
  uint32_t batch_size = (uint32_t)atoi(argv[3]);
  uint32_t num_layers = (uint32_t)(argc - 4);
  
  // Training Dataset Initialization 
  uint8_t* train_images_raw = read_binary_file("data/train-images-idx3-ubyte");
  uint8_t* train_labels_raw = read_binary_file("data/train-labels-idx1-ubyte");
  
  mnist_image_file_header_t train_images_header = *((mnist_image_file_header_t*)train_images_raw);
  images_header_to_little_endian(&train_images_header); 
  if(train_images_header.magic_number != MNIST_IMAGE_MAGIC) {
    printf("Train image file magic (checksum) doesn't match.\n");
    exit(-1);
  }

  mnist_label_file_header_t train_labels_header = *((mnist_label_file_header_t*)train_labels_raw);
  labels_header_to_little_endian(&train_labels_header);
  if(train_labels_header.magic_number != MNIST_LABEL_MAGIC) {
    printf("Train label file magic (checksum) doesn't match.\n");
    exit(-1);
  }

  if(train_images_header.num_images != train_labels_header.num_labels) {
    printf("Number of images has to match number of labels!\n");
    exit(-1);
  }

  mnist_dataset_t train_dataset;
  init_dataset(&train_dataset, &train_images_header, train_images_raw, &train_labels_header, train_labels_raw, batch_size);
  free(train_images_raw);
  free(train_labels_raw);

  // Test Dataset Initialization
  uint8_t* test_images_raw = read_binary_file("data/t10k-images-idx3-ubyte");
  uint8_t* test_labels_raw = read_binary_file("data/t10k-labels-idx1-ubyte");
  
  mnist_image_file_header_t test_images_header = *((mnist_image_file_header_t*)test_images_raw);
  images_header_to_little_endian(&test_images_header); 
  if(test_images_header.magic_number != MNIST_IMAGE_MAGIC) {
    printf("Train image file magic (checksum) doesn't match.\n");
    exit(-1);
  }

  mnist_label_file_header_t test_labels_header = *((mnist_label_file_header_t*)test_labels_raw);
  labels_header_to_little_endian(&test_labels_header);
  if(test_labels_header.magic_number != MNIST_LABEL_MAGIC) {
    printf("Train label file magic (checksum) doesn't match.\n");
    exit(-1);
  }

  if(test_images_header.num_images != test_labels_header.num_labels) {
    printf("Number of images has to match number of labels!\n");
    exit(-1);
  }

  mnist_dataset_t test_dataset;
  init_dataset(&test_dataset, &test_images_header, test_images_raw, &test_labels_header, test_labels_raw, batch_size);
  free(test_images_raw);
  free(test_labels_raw);
  
  layer_dims_arr = (uint32_t*)malloc(num_layers * sizeof(uint32_t));
  for(i=0; i<num_layers; i++) {
    layer_dims_arr[i] = (uint32_t)atoi(argv[i + 4]);
  }
  if(layer_dims_arr[0] != train_images_header.num_rows * train_images_header.num_cols) {
    printf("Number of nodes in first layer must match pixel count of each image in dataset.\n");
    exit(-1);
  }
  
  hipSetDevice(gpunum);
  printf("Using device %d\n", gpunum);
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);

  layer_metadata_t* layer_mtdt_arr = init_layer_metadata_arr(layer_dims_arr, num_layers, batch_size);
  
  // TRAIN
  for(i=0; i<epochs; i++) {
    printf("\n\n******EPOCH %d******\n\n", i+1);
    train_mnist(&train_dataset, layer_mtdt_arr, layer_dims_arr, num_layers, learning_rate);
  }

  // TEST
  printf("\n\n******TEST******\n\n");
  test_mnist(&test_dataset, layer_mtdt_arr, layer_dims_arr, num_layers);
  
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop);  

  printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time
  
  free(layer_dims_arr);
  free_layer_metadata_arr(layer_mtdt_arr, num_layers);
  free_dataset(&train_dataset);
  free_dataset(&test_dataset);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
