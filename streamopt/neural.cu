#define FP float

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include <stdbool.h>
#include <omp.h>

#define MNIST_LABEL_MAGIC 0x00000801
#define MNIST_IMAGE_MAGIC 0x00000803
#define MNIST_LABELS 10
#define BLOCK_SZ 32
#define NUM_STREAMS 6

typedef struct mnist_label_file_header_t_ {
  uint32_t magic_number;
  uint32_t num_labels;
} __attribute__((packed)) mnist_label_file_header_t;  

typedef struct mnist_image_file_header_t_ {
  uint32_t magic_number;
  uint32_t num_images;
  uint32_t num_rows;
  uint32_t num_cols;
} __attribute__((packed)) mnist_image_file_header_t;

typedef struct mnist_batch_t_ {
  FP* pixels;
  FP* labels;
  uint32_t size;
} __attribute__((packed)) mnist_batch_t;

typedef struct mnist_dataset_t_ {
  mnist_batch_t* batches;
  uint32_t num_batches;
} mnist_dataset_t;

typedef struct layer_metadata_t_ {
  FP* weights; // malloced in device (contains bias)
  FP* deltas;
  FP* outputs;
  FP* activations; // malloced in device
  uint32_t n; // number of rows in A
  uint32_t p; // number of cols in A, number of rows in B
} layer_metadata_t;

typedef struct backprop_args_t_ {
  FP* d_curr_derivative;
  FP* d_next_derivative;
} backprop_args_t;

typedef struct activation_deriv_args_t_ {
  FP* d_softmax_jacobian;
  FP* d_layer_output_single;
  FP* d_softmax_derivative;
  FP* d_max_val;
  FP* d_sum;
  FP* d_relu_derivative;
} activation_deriv_args_t;

typedef struct backprop_params_t_ {
  FP* d_xentropy_derivative;
  int idx_in_batch;
  uint32_t batch_size;
  layer_metadata_t* layer_mtdt_arr;
  uint32_t* layer_dims_arr;
  uint32_t num_layers;
  FP learning_rate;
  hipStream_t* streams;
  backprop_args_t* backprop_args_arr;
  activation_deriv_args_t* actv_deriv_args_arr;
  FP** d_activation_deriv_arr;
} backprop_params_t;

typedef struct cat_xentropy_args_t_ {
  FP* d_max_arr;
  FP* d_sum_arr;
  FP* xentropy_arr;
  FP* d_xentropy_arr;
} cat_xentropy_args_t;

__device__ static float atomicMax(float* address, float val) {
  int* address_as_i = (int*) address;
  int old = *address_as_i, assumed;
  do {
    assumed = old;
    old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
  } while (assumed != old);
  return __int_as_float(old);
}

__global__ void blocked_gpu_matrixmult(FP* A, FP* B, FP* C, FP* C_activations, int n, int m, int p, bool forward_pass_flag) {
  // position within submatrix
  int C_col = threadIdx.x;
  int C_row = threadIdx.y;
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
  // A_row_abs = C_row_abs
  int A_col_abs = C_col;
  int B_row_abs = C_row;
  // B_col_abs = C_col_abs

  // total number of submatrixes
  int totalSub = (p + BLOCK_SZ - 1) / BLOCK_SZ;
  int idxSub;
  int k;
  FP Cval = 0;
 
  for(idxSub = 0; idxSub < totalSub; idxSub++) {
    // Must use fixed block size as dynamic allocation is not possible
    __shared__ FP A_shr[BLOCK_SZ][BLOCK_SZ];
    __shared__ FP B_shr[BLOCK_SZ][BLOCK_SZ];
    if(C_row_abs < n && A_col_abs < p) {
      A_shr[C_row][C_col] = A[C_row_abs * p + A_col_abs];
    }
    if(B_row_abs < p && C_col_abs < m) {
      if(forward_pass_flag) {
        if(B_row_abs == 0) {
          // Bias applied
          B_shr[C_row][C_col] = 1;
        }
        else {
          B_shr[C_row][C_col] = B[(B_row_abs - 1) * m + C_col_abs];
        }
      }
      else
        B_shr[C_row][C_col] = B[B_row_abs * m + C_col_abs];
    }
    // Synchronize
    __syncthreads();
    if(C_row_abs < n && C_col_abs < m) {
      // Multiply A submatrix and B submatrix
      if(idxSub == (totalSub - 1) && p % BLOCK_SZ != 0) {
        for(k = 0; k < p % BLOCK_SZ; k++)
          Cval += A_shr[C_row][k] * B_shr[k][C_col];
      }
      else {
        for(k = 0; k < BLOCK_SZ; k++)
          Cval += A_shr[C_row][k] * B_shr[k][C_col];
      }
    }
    // Synchronize
    __syncthreads();
    A_col_abs += BLOCK_SZ;
    B_row_abs += BLOCK_SZ;
  }
  if(C_row_abs < n && C_col_abs < m) {
    C[C_row_abs * m + C_col_abs] = Cval;
    if(forward_pass_flag) {
      // ReLU activation applied
      C_activations[C_row_abs * m + C_col_abs] = (Cval > 0) ? Cval : 0;
    }
  }
}

FP randn(double mu, double sigma) {
  double U1, U2, W, mult;
  static double X1, X2;
  static int call = 0;
 
  if (call == 1) {
    call = !call;
    return (mu + sigma * (double) X2);
  }
 
  do {
    U1 = -1 + ((double) rand () / RAND_MAX) * 2;
    U2 = -1 + ((double) rand () / RAND_MAX) * 2;
    W = pow (U1, 2) + pow (U2, 2);
  }
  while (W >= 1 || W == 0);
 
  mult = sqrt ((-2 * log (W)) / W);
  X1 = U1 * mult;
  X2 = U2 * mult;
 
  call = !call;
 
  return (FP)(mu + sigma * (double) X1);
}

uint8_t* read_binary_file(const char* filename) {
  FILE *fileptr;
  uint8_t *buffer;
  long filelen;

  fileptr = fopen(filename, "rb");
  fseek(fileptr, 0, SEEK_END);
  filelen = ftell(fileptr);
  rewind(fileptr);

  buffer = (uint8_t *)malloc(filelen * sizeof(uint8_t));
  fread(buffer, filelen, 1, fileptr);
  fclose(fileptr);
  return buffer;
}

uint32_t big_to_little_endian(uint32_t num) {
  return (
    ((num & 0xFF000000) >> 24) |
    ((num & 0x00FF0000) >>  8) |
    ((num & 0x0000FF00) <<  8) |
    ((num & 0x000000FF) << 24)
  );
}

void images_header_to_little_endian(mnist_image_file_header_t* images_header) {
  images_header->magic_number = big_to_little_endian(images_header->magic_number);
  images_header->num_images = big_to_little_endian(images_header->num_images);
  images_header->num_rows = big_to_little_endian(images_header->num_rows);
  images_header->num_cols = big_to_little_endian(images_header->num_cols);
}

void labels_header_to_little_endian(mnist_label_file_header_t* labels_header) {
  labels_header->magic_number = big_to_little_endian(labels_header->magic_number);
  labels_header->num_labels = big_to_little_endian(labels_header->num_labels);

}

void init_dataset(mnist_dataset_t* dataset, mnist_image_file_header_t* images_header, uint8_t* images_raw, mnist_label_file_header_t* labels_header, uint8_t* labels_raw, uint32_t batch_size) {
  if(images_header->num_images % batch_size == 0)
    dataset->num_batches = images_header->num_images / batch_size;
  else
    dataset->num_batches = images_header->num_images / batch_size + 1;
  dataset->batches = (mnist_batch_t*)malloc(dataset->num_batches * sizeof(mnist_batch_t));
  uint64_t image_total_pixels = images_header->num_rows * images_header->num_cols;
  uint64_t image_dataset_total_bytes = sizeof(mnist_image_file_header_t) + image_total_pixels * images_header->num_images;
  uint64_t actual_batch_size;
  int batch_idx = 0;
  int pixel_idx, label_idx;
  int raw_label_idx = sizeof(mnist_label_file_header_t);
  uint64_t i, j, k;

  for(i=sizeof(mnist_image_file_header_t); i<image_dataset_total_bytes; i+=image_total_pixels * batch_size) {
    if(i + image_total_pixels * batch_size > image_dataset_total_bytes)
      actual_batch_size = (image_dataset_total_bytes - i) / image_total_pixels;
    else
      actual_batch_size = batch_size;
    dataset->batches[batch_idx].pixels = (FP*)malloc(actual_batch_size * image_total_pixels * sizeof(FP));
    dataset->batches[batch_idx].labels = (FP*)malloc(actual_batch_size * MNIST_LABELS * sizeof(FP));
    dataset->batches[batch_idx].size = actual_batch_size;
    pixel_idx = 0;
    for(j=i; j<i + image_total_pixels; j++) {
      for(k=j; k<i + image_total_pixels * actual_batch_size; k+=image_total_pixels) {
        dataset->batches[batch_idx].pixels[pixel_idx] = images_raw[k] / (FP)255;
        pixel_idx++;
      }
    }
    label_idx = 0;
    for(k=0; k<MNIST_LABELS; k++) {
      for(j=raw_label_idx; j<raw_label_idx + actual_batch_size; j++) {
        dataset->batches[batch_idx].labels[label_idx] = labels_raw[j] == k;
        label_idx++;
      }
    }
    raw_label_idx += actual_batch_size; 
    batch_idx++;
  }
}

void free_dataset(mnist_dataset_t* dataset) {
  int batch_idx;
  for(batch_idx=0; batch_idx<dataset->num_batches; batch_idx++) {
    free(dataset->batches[batch_idx].pixels);
    free(dataset->batches[batch_idx].labels);
  }
  free(dataset->batches);
}

layer_metadata_t* init_layer_metadata_arr(uint32_t* layer_dims_arr, uint32_t num_layers, uint32_t batch_size) {
  layer_metadata_t* layer_mtdt_arr = (layer_metadata_t*)malloc(num_layers * sizeof(layer_metadata_t));
  uint32_t prev_dim, curr_dim;
  int i, k;

  layer_mtdt_arr[0].weights = NULL;
  layer_mtdt_arr[0].deltas = NULL;
  layer_mtdt_arr[0].outputs = NULL;
  hipMalloc((void**)&layer_mtdt_arr[0].activations, layer_dims_arr[0] * batch_size * sizeof(FP));

  for(i=1; i<num_layers; i++) {
    prev_dim = layer_dims_arr[i-1];
    curr_dim = layer_dims_arr[i];
    hipMalloc((void**)&layer_mtdt_arr[i].weights, (prev_dim + 1) * curr_dim * sizeof(FP));
    FP* init_weights = (FP*)malloc((prev_dim + 1) * curr_dim * sizeof(FP));
    for(k=0; k<(prev_dim + 1) * curr_dim; k++) {
      if(k % (prev_dim + 1) == 0) {
        init_weights[k] = 0;
        continue;
      }
      init_weights[k] = randn(0, sqrt(2/(double) prev_dim));
    }
    hipMemcpy(layer_mtdt_arr[i].weights, init_weights, (prev_dim + 1) * curr_dim * sizeof(FP), hipMemcpyHostToDevice);
    free(init_weights);
    hipMalloc((void**)&layer_mtdt_arr[i].deltas, (prev_dim + 1) * curr_dim * sizeof(FP));
    FP* init_deltas = (FP*)malloc((prev_dim + 1) * curr_dim * sizeof(FP));
    for(k=0; k<(prev_dim + 1) * curr_dim; k++) {
      init_deltas[k] = 0;
    }
    hipMemcpy(layer_mtdt_arr[i].deltas, init_deltas, (prev_dim + 1) * curr_dim * sizeof(FP), hipMemcpyHostToDevice);
    free(init_deltas);
    hipMalloc((void**)&layer_mtdt_arr[i].outputs, curr_dim * batch_size * sizeof(FP));
    hipMalloc((void**)&layer_mtdt_arr[i].activations, curr_dim * batch_size * sizeof(FP));
    layer_mtdt_arr[i].n = curr_dim;
    layer_mtdt_arr[i].p = prev_dim + 1;
  }
  
  return layer_mtdt_arr;
}

void free_layer_metadata_arr(layer_metadata_t* layer_mtdt_arr, uint32_t num_layers) {
  int i;
  hipFree(layer_mtdt_arr[0].activations);
  for(i=1; i<num_layers; i++) {
    hipFree(layer_mtdt_arr[i].weights);
    hipFree(layer_mtdt_arr[i].deltas);
    hipFree(layer_mtdt_arr[i].outputs);
    hipFree(layer_mtdt_arr[i].activations);
  }
  free(layer_mtdt_arr);
}

__global__ void calc_max_output_layer(FP* C, int n, int m, FP* d_max_arr) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
 
  if(C_row_abs < n && C_col_abs < m)
    atomicMax(&d_max_arr[C_col_abs], C[C_row_abs * m + C_col_abs]);
}

__global__ void calc_sum_output_layer(FP* C, int n, int m, FP* d_max_arr, FP* d_sum_arr) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
 
  if(C_row_abs < n && C_col_abs < m)
    atomicAdd(&d_sum_arr[C_col_abs], ::expf(C[C_row_abs * m + C_col_abs] - d_max_arr[C_col_abs])); 
}

__global__ void softmax_output_layer(FP* C, FP* C_activations, int n, int m, FP* d_max_arr, FP* d_sum_arr, FP* d_xentropy_arr) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
 
  if(C_row_abs < n && C_col_abs < m) {
    C_activations[C_row_abs * m + C_col_abs] = ::expf(C[C_row_abs * m + C_col_abs] - d_max_arr[C_col_abs] - ::logf(d_sum_arr[C_col_abs]));
    d_xentropy_arr[C_col_abs] = 0;
  }
}

__global__ void crossentropy_output_layer(FP* C_activations, int n, int m, FP* d_labels, FP* d_max_arr, FP* d_sum_arr, FP* d_xentropy_arr) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
 
  if(C_row_abs < n && C_col_abs < m) {
    d_max_arr[C_col_abs] = -INFINITY;
    d_sum_arr[C_col_abs] = 0;
    atomicAdd(&d_xentropy_arr[C_col_abs], -1 * d_labels[C_row_abs * m + C_col_abs] * ::logf(C_activations[C_row_abs * m + C_col_abs]));
  }
}

FP* calc_categorical_xentropy(FP* output_layer_outputs, FP* output_layer_activations, uint32_t batch_size, FP* d_labels, cat_xentropy_args_t* args) {
  FP* d_max_arr = args->d_max_arr;
  FP* d_sum_arr = args->d_sum_arr;
  FP* xentropy_arr = args->xentropy_arr;
  FP* d_xentropy_arr = args->d_xentropy_arr;
  
  dim3 Grid;
  Grid.x = (batch_size + BLOCK_SZ - 1) / BLOCK_SZ;
  Grid.y = (MNIST_LABELS + BLOCK_SZ - 1) / BLOCK_SZ; 
  dim3 Block(BLOCK_SZ, BLOCK_SZ);

  calc_max_output_layer<<<Grid,Block>>>(output_layer_outputs, MNIST_LABELS, batch_size, d_max_arr);
  calc_sum_output_layer<<<Grid,Block>>>(output_layer_outputs, MNIST_LABELS, batch_size, d_max_arr, d_sum_arr);
  softmax_output_layer<<<Grid,Block>>>(output_layer_outputs, output_layer_activations, MNIST_LABELS, batch_size, d_max_arr, d_sum_arr, d_xentropy_arr);
  
  crossentropy_output_layer<<<Grid,Block>>>(output_layer_activations, MNIST_LABELS, batch_size, d_labels, d_max_arr, d_sum_arr, d_xentropy_arr);

  hipMemcpy(xentropy_arr, d_xentropy_arr, batch_size * sizeof(FP), hipMemcpyDeviceToHost);

  return xentropy_arr;
}

__global__ void extract_single_example(FP* C, int n, int m, FP* C_full, int batch_size, int idx_in_batch, FP* d_max_val, FP* d_sum) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
 
  if(C_row_abs < n && C_col_abs < m) {
    C[C_row_abs] = C_full[C_row_abs * batch_size + idx_in_batch];
    if(d_max_val != NULL) *d_max_val = -INFINITY;
    if(d_sum != NULL) *d_sum = 0;
  }
}

__global__ void calc_single_output_max(FP* C, int n, int m, FP* d_max_val) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
 
  if(C_row_abs < n && C_col_abs < m)
    atomicMax(d_max_val, C[C_row_abs * m + C_col_abs]);
}

__global__ void calc_single_output_sum(FP* C, int n, int m, FP* d_max_val, FP* d_sum) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
 
  if(C_row_abs < n && C_col_abs < m)
    atomicAdd(d_sum, ::expf(C[C_row_abs] - *d_max_val));
}

__global__ void calc_softmax_jacobian(FP* C, int n, int m, FP* d_layer_output_single, FP* d_max_val, FP* d_sum) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
 
  if(C_row_abs < n && C_col_abs < m) {
    FP softmax_output_i = ::expf(d_layer_output_single[C_row_abs] - *d_max_val - ::logf(*d_sum));
    if(C_row_abs == C_col_abs)
      C[C_row_abs * m + C_col_abs] = softmax_output_i * (1 - softmax_output_i);
    else {
      FP softmax_output_j = ::expf(d_layer_output_single[C_col_abs] - *d_max_val - ::logf(*d_sum));
      C[C_row_abs * m + C_col_abs] = -1 * softmax_output_i * softmax_output_j;
    }
  }
}

FP* calc_softmax_derivative(FP* d_layer_outputs, int idx_in_batch, uint32_t batch_size, uint32_t layer_dim, hipStream_t* streams, activation_deriv_args_t* args_arr) {
  uint32_t curr_thread = idx_in_batch % NUM_STREAMS;
  int k;
  
  dim3 Grid;
  dim3 Block(BLOCK_SZ, BLOCK_SZ);
  Grid.x = (1 + BLOCK_SZ - 1) / BLOCK_SZ;
  Grid.y = (layer_dim + BLOCK_SZ - 1) / BLOCK_SZ;

  if(curr_thread == 0) {
    for(k=0; k<NUM_STREAMS; k++) {
      if(idx_in_batch + k >= batch_size) break;
      extract_single_example<<<Grid,Block,0,streams[k]>>>(args_arr[k].d_layer_output_single, layer_dim, 1, d_layer_outputs, batch_size, idx_in_batch + k, args_arr[k].d_max_val, args_arr[k].d_sum);
    }
    for(k=0; k<NUM_STREAMS; k++) {
      if(idx_in_batch + k >= batch_size) break;
      calc_single_output_max<<<Grid,Block,0,streams[k]>>>(args_arr[k].d_layer_output_single, layer_dim, 1, args_arr[k].d_max_val);
    }
    for(k=0; k<NUM_STREAMS; k++) {
      if(idx_in_batch + k >= batch_size) break;
      calc_single_output_sum<<<Grid,Block,0,streams[k]>>>(args_arr[k].d_layer_output_single, layer_dim, 1, args_arr[k].d_max_val, args_arr[k].d_sum);
    }
    Grid.x = (layer_dim + BLOCK_SZ - 1) / BLOCK_SZ;
    Grid.y = (layer_dim + BLOCK_SZ - 1) / BLOCK_SZ;
    for(k=0; k<NUM_STREAMS; k++) {
      if(idx_in_batch + k >= batch_size) break;
      calc_softmax_jacobian<<<Grid,Block,0,streams[k]>>>(args_arr[k].d_softmax_jacobian, layer_dim, layer_dim, args_arr[k].d_layer_output_single, args_arr[k].d_max_val, args_arr[k].d_sum);
    }
    Grid.x = (layer_dim + BLOCK_SZ - 1) / BLOCK_SZ;
    Grid.y = (1 + BLOCK_SZ - 1) / BLOCK_SZ;
    for(k=0; k<NUM_STREAMS; k++) {
      if(idx_in_batch + k >= batch_size) break;
      blocked_gpu_matrixmult<<<Grid,Block,0,streams[k]>>>(args_arr[k].d_layer_output_single, args_arr[k].d_softmax_jacobian, args_arr[k].d_softmax_derivative, NULL, 1, layer_dim, layer_dim, false);
    }
  }
  #pragma omp barrier

  return args_arr[curr_thread].d_softmax_derivative;
}

__global__ void calc_relu_derivative_gpu(FP* C, int n, int m, FP* d_layer_output_single) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
 
  if(C_row_abs < n && C_col_abs < m)
    C[C_row_abs] = (d_layer_output_single[C_row_abs] > 0) ? 1 : 0;
}

FP* calc_relu_derivative(FP* d_layer_outputs, int idx_in_batch, uint32_t batch_size, uint32_t layer_dim, hipStream_t* streams, activation_deriv_args_t* args_arr) {
  uint32_t curr_thread = idx_in_batch % NUM_STREAMS;
  int k;

  dim3 Grid;
  dim3 Block(BLOCK_SZ, BLOCK_SZ);
  Grid.x = (1 + BLOCK_SZ - 1) / BLOCK_SZ;
  Grid.y = (layer_dim + BLOCK_SZ - 1) / BLOCK_SZ;

  if(curr_thread == 0) {
    for(k=0; k<NUM_STREAMS; k++) {
      if(idx_in_batch + k >= batch_size) break;
      extract_single_example<<<Grid,Block,0,streams[k]>>>(args_arr[k].d_layer_output_single, layer_dim, 1, d_layer_outputs, batch_size, idx_in_batch + k, NULL, NULL);
    }
    for(k=0; k<NUM_STREAMS; k++) {
      if(idx_in_batch + k >= batch_size) break;
      calc_relu_derivative_gpu<<<Grid,Block,0,streams[k]>>>(args_arr[k].d_relu_derivative, layer_dim, 1, args_arr[k].d_layer_output_single);
    }
  }
  #pragma omp barrier

  return args_arr[curr_thread].d_relu_derivative;
}

__global__ void update_deltas_gpu(FP* curr_derivative, FP* activations, int idx_in_batch, int batch_size, FP* deltas, int n, int m, FP learning_rate) {
  // position within submatrix
  int C_col = threadIdx.x;
  int C_row = threadIdx.y;
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
  FP Cval;
 
  if(C_row_abs < n && C_col_abs < m) {
    // Must use fixed block size as dynamic allocation is not possible
    __shared__ FP actv_shr[BLOCK_SZ];
    __shared__ FP deriv_shr[BLOCK_SZ];
    if(C_row == 0) {
      int actv_idx = (C_col_abs - 1) * batch_size + idx_in_batch;
      actv_shr[C_col] = (C_col_abs == 0) ? 1 : activations[actv_idx];
    }
    if(C_col == 0)
      deriv_shr[C_row] = curr_derivative[C_row_abs];
    
    // Synchronize
    __syncthreads();

    Cval = learning_rate * deriv_shr[C_row] * actv_shr[C_col];
    atomicAdd(&deltas[C_row_abs * m + C_col_abs], ::fdividef(Cval, batch_size));
  }
}

void update_deltas(backprop_args_t* backprop_args_arr, FP* d_activations, int idx_in_batch, int batch_size, FP* d_deltas, int n, int m, FP learning_rate, hipStream_t* streams) {
  uint32_t curr_thread = idx_in_batch % NUM_STREAMS;
  int k;

  dim3 Block(BLOCK_SZ, BLOCK_SZ);
  dim3 Grid;
  Grid.x = (m + BLOCK_SZ - 1) / BLOCK_SZ;
  Grid.y = (n + BLOCK_SZ - 1) / BLOCK_SZ;
  
  if(curr_thread == 0) {
    for(k=0; k<NUM_STREAMS; k++) {
      if(idx_in_batch + k >= batch_size) break;
      update_deltas_gpu<<<Grid,Block,0,streams[k]>>>(backprop_args_arr[k].d_curr_derivative, d_activations, idx_in_batch + k, batch_size, d_deltas, n, m, learning_rate);
    }
  }
  #pragma omp barrier
}

__global__ void calc_hadamard_product(FP* d_curr_derivative, int n, int m, FP* d_next_derivative, FP* d_activation_derivative) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
 
  if(C_row_abs < n && C_col_abs < m)
    d_curr_derivative[C_col_abs] = d_next_derivative[C_col_abs + 1] * d_activation_derivative[C_col_abs];
}

void stochastic_backprop(void* backprop_params) {
  backprop_params_t* params = (backprop_params_t*)backprop_params;
  FP* d_xentropy_derivative = params->d_xentropy_derivative;
  int idx_in_batch = params->idx_in_batch;
  uint32_t batch_size = params->batch_size;
  layer_metadata_t* layer_mtdt_arr = params->layer_mtdt_arr;
  uint32_t* layer_dims_arr = params->layer_dims_arr;
  uint32_t num_layers = params->num_layers;
  FP learning_rate = params->learning_rate;
  hipStream_t* streams = params->streams;
  backprop_args_t* args_arr = params->backprop_args_arr;
  activation_deriv_args_t* actv_deriv_args_arr = params->actv_deriv_args_arr;
  FP** d_activation_deriv_arr = params->d_activation_deriv_arr;

  uint32_t curr_thread = idx_in_batch % NUM_STREAMS;
  int i, k;

  // Extract the cross entropy derivative for a specific observation in batch based on idx_in_batch
  dim3 Block(BLOCK_SZ, BLOCK_SZ);
  dim3 Grid;  
  Grid.x = (1 + BLOCK_SZ - 1) / BLOCK_SZ;
  Grid.y = (MNIST_LABELS + BLOCK_SZ - 1) / BLOCK_SZ;
  if(curr_thread == 0) {
    for(k=0; k<NUM_STREAMS; k++) {
      if(idx_in_batch + k >= batch_size) break;
      extract_single_example<<<Grid,Block,0,streams[k]>>>(args_arr[k].d_curr_derivative, MNIST_LABELS, 1, d_xentropy_derivative, batch_size, idx_in_batch + k, NULL, NULL);
    }
  }
  #pragma omp barrier

  uint32_t curr_dim, next_dim;
  for(i=num_layers - 1; i>=2; i--) {
    curr_dim = layer_dims_arr[i];
    next_dim = layer_dims_arr[i - 1];

    if(i == num_layers - 1)
      d_activation_deriv_arr[curr_thread] = calc_softmax_derivative(layer_mtdt_arr[i - 1].outputs, idx_in_batch, batch_size, next_dim, streams, actv_deriv_args_arr);
    else
      d_activation_deriv_arr[curr_thread] = calc_relu_derivative(layer_mtdt_arr[i - 1].outputs, idx_in_batch, batch_size, next_dim, streams, actv_deriv_args_arr);
    
    Grid.x = ((next_dim + 1) + BLOCK_SZ - 1) / BLOCK_SZ;
    Grid.y = (1 + BLOCK_SZ - 1) / BLOCK_SZ;
    if(curr_thread == 0) {
      for(k=0; k<NUM_STREAMS; k++) {
        if(idx_in_batch + k >= batch_size) break;
        blocked_gpu_matrixmult<<<Grid,Block,0,streams[k]>>>(args_arr[k].d_curr_derivative, layer_mtdt_arr[i].weights, args_arr[k].d_next_derivative, NULL, 1, next_dim + 1, curr_dim, false);
      }
    }
    #pragma omp barrier
    
    update_deltas(args_arr, layer_mtdt_arr[i - 1].activations, idx_in_batch, batch_size, layer_mtdt_arr[i].deltas, curr_dim, next_dim + 1, learning_rate, streams);

    Grid.x = (next_dim + BLOCK_SZ - 1) / BLOCK_SZ;
    if(curr_thread == 0) {
      for(k=0; k<NUM_STREAMS; k++) {
        if(idx_in_batch + k >= batch_size) break;
        calc_hadamard_product<<<Grid,Block,0,streams[k]>>>(args_arr[k].d_curr_derivative, 1, next_dim, args_arr[k].d_next_derivative, d_activation_deriv_arr[k]);
      }
    }
    #pragma omp barrier
  }

  update_deltas(args_arr, layer_mtdt_arr[0].activations, idx_in_batch, batch_size, layer_mtdt_arr[1].deltas, layer_dims_arr[1], layer_dims_arr[0] + 1, learning_rate, streams);
}

void exec_stochastic_backprop(uint32_t curr_thread, void* backprop_params) {
  backprop_params_t* params = (backprop_params_t*)backprop_params;
  uint32_t batch_size = params->batch_size;
  int j;

  for(j=0; j<batch_size; j+=NUM_STREAMS) {
    params->idx_in_batch = j + curr_thread;
    stochastic_backprop(backprop_params);
  }
}

__global__ void update_weights_biases(FP* weights, FP* deltas, FP batch_size, int n, int m) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
 
  if(C_row_abs < n && C_col_abs < m) {
    weights[C_row_abs * m + C_col_abs] -= deltas[C_row_abs * m + C_col_abs];
    deltas[C_row_abs * m + C_col_abs] = 0;
  }
}

FP calc_accuracy(FP* activations, FP* labels, uint32_t batch_size, uint32_t* correct_cnt_arr) {
  int num_correct = 0;
  int i, j;
  int actv_idx;
  FP max_val;
  int max_i;
  for(j=0; j<batch_size; j++) {
    actv_idx = j;
    max_val = -INFINITY;
    for(i=0; i<MNIST_LABELS; i++) {
      if(activations[actv_idx] > max_val) {
        max_val = activations[actv_idx];
        max_i = i;
      }
      actv_idx += batch_size;
    }
    if((int)round(labels[max_i * batch_size + j]) == 1) {
      num_correct++;
      if(correct_cnt_arr != NULL)
        correct_cnt_arr[max_i]++;
    }
  }
  return num_correct / (FP)batch_size;
}

__global__ void calc_xentropy_derivative(FP* C, int n, int m, FP* d_labels, FP* derivative) {
  // absolute position
  int C_col_abs = threadIdx.x + BLOCK_SZ * blockIdx.x;
  int C_row_abs = threadIdx.y + BLOCK_SZ * blockIdx.y;
 
  if(C_row_abs < n && C_col_abs < m)
    derivative[C_row_abs * m + C_col_abs] = C[C_row_abs * m + C_col_abs] - d_labels[C_row_abs * m + C_col_abs];
}

void mini_batch_gradient_descent(mnist_dataset_t* train_dataset, layer_metadata_t* layer_mtdt_arr, uint32_t* layer_dims_arr,
  uint32_t num_layers, FP learning_rate, FP* d_labels, cat_xentropy_args_t* xentropy_args, FP* output_layer_activations,
  FP* d_xentropy_derivative, backprop_params_t* backprop_params_arr, hipStream_t* streams, backprop_args_t* backprop_args_arr,
  activation_deriv_args_t* actv_deriv_args_arr, FP** d_activation_deriv_arr, uint32_t curr_thread) {

  backprop_params_arr[curr_thread].d_xentropy_derivative = d_xentropy_derivative;  
  backprop_params_arr[curr_thread].layer_mtdt_arr = layer_mtdt_arr;
  backprop_params_arr[curr_thread].layer_dims_arr = layer_dims_arr;
  backprop_params_arr[curr_thread].num_layers = num_layers;
  backprop_params_arr[curr_thread].learning_rate = learning_rate;
  backprop_params_arr[curr_thread].streams = streams;
  backprop_params_arr[curr_thread].backprop_args_arr = backprop_args_arr;
  backprop_params_arr[curr_thread].actv_deriv_args_arr = actv_deriv_args_arr;
  backprop_params_arr[curr_thread].d_activation_deriv_arr = d_activation_deriv_arr;

  int i, j;
  uint32_t actual_batch_size;
  dim3 Grid;
  dim3 Block(BLOCK_SZ, BLOCK_SZ);
  FP* xentropy_arr;
  FP batch_train_accuracy;
  
  // Mini Batch Gradient Descent
  for(i=0; i<train_dataset->num_batches; i++) {
    actual_batch_size = train_dataset->batches[i].size; 

    // PERFORMED BY ONE THREAD
    if(curr_thread == 0) {
      hipMemcpy(layer_mtdt_arr[0].activations, train_dataset->batches[i].pixels, layer_dims_arr[0] * actual_batch_size * sizeof(FP), hipMemcpyHostToDevice);
      for(j=1; j<num_layers; j++) {
        Grid.x = (actual_batch_size + BLOCK_SZ - 1) / BLOCK_SZ;
        Grid.y = (layer_mtdt_arr[j].n + BLOCK_SZ - 1) / BLOCK_SZ;
        // Calculate outputs and ReLU activations for next layer
        blocked_gpu_matrixmult<<<Grid,Block>>>(layer_mtdt_arr[j].weights, layer_mtdt_arr[j-1].activations, layer_mtdt_arr[j].outputs, layer_mtdt_arr[j].activations, layer_mtdt_arr[j].n, actual_batch_size, layer_mtdt_arr[j].p, true);
      }
      
      // Apply softmax to last layer and calculate categorical cross entropy loss
      hipMemcpy(d_labels, train_dataset->batches[i].labels, MNIST_LABELS * actual_batch_size * sizeof(FP), hipMemcpyHostToDevice);
      xentropy_arr = calc_categorical_xentropy(layer_mtdt_arr[num_layers - 1].outputs, layer_mtdt_arr[num_layers - 1].activations, actual_batch_size, d_labels, xentropy_args);
      
      // Calculate training accuracy for batch
      hipMemcpy(output_layer_activations, layer_mtdt_arr[num_layers - 1].activations, MNIST_LABELS * actual_batch_size * sizeof(FP), hipMemcpyDeviceToHost);
      batch_train_accuracy = calc_accuracy(output_layer_activations, train_dataset->batches[i].labels, actual_batch_size, NULL);
      printf("Batch %d Train Accuracy: %.2f\n", i + 1, batch_train_accuracy);
      
      // Calculate cross entropy derivative (apply softmax and compare with labels)
      Grid.x = (actual_batch_size + BLOCK_SZ - 1) / BLOCK_SZ;
      Grid.y = (MNIST_LABELS + BLOCK_SZ - 1) / BLOCK_SZ;
      calc_xentropy_derivative<<<Grid,Block>>>(layer_mtdt_arr[num_layers - 1].activations, MNIST_LABELS, actual_batch_size, d_labels, d_xentropy_derivative);
    }
    #pragma omp barrier

    // Stochastic backpropagation for each example in batch
    backprop_params_arr[curr_thread].batch_size = actual_batch_size;
    exec_stochastic_backprop(curr_thread, (void*)&backprop_params_arr[curr_thread]);
    #pragma omp barrier

    // PERFORMED BY ONE THREAD
    if(curr_thread == 0) {
      // Update weights and biases with computed average deltas
      for(j=1; j<num_layers; j++) {
        Grid.x = ((layer_dims_arr[j - 1] + 1) + BLOCK_SZ - 1) / BLOCK_SZ;
        Grid.y = (layer_dims_arr[j] + BLOCK_SZ - 1) / BLOCK_SZ;
        update_weights_biases<<<Grid,Block>>>(layer_mtdt_arr[j].weights, layer_mtdt_arr[j].deltas, (FP)actual_batch_size, layer_dims_arr[j], layer_dims_arr[j - 1] + 1);
      }
      // PRINT START
      /*printf("\n\nBatch %d Cross Entropy\n\n", i);
      for(j=0; j<actual_batch_size; j++) {
        printf("%.2f ", xentropy_arr[j]);
      }
      printf("\n");*/
      // PRINT END
    }
    #pragma omp barrier
  }
}

void train_mnist(mnist_dataset_t* train_dataset, layer_metadata_t* layer_mtdt_arr, uint32_t* layer_dims_arr, uint32_t num_layers, FP learning_rate) {
  int i, k;
  FP* output_layer_activations = (FP*)malloc(MNIST_LABELS * train_dataset->batches[0].size * sizeof(FP));
  backprop_params_t* backprop_params_arr = (backprop_params_t*)malloc(sizeof(backprop_params_t) * NUM_STREAMS);
  hipStream_t streams[NUM_STREAMS];
  for(i=0; i<NUM_STREAMS; i++) {
    hipStreamCreate(&streams[i]);
  }
  FP* d_labels;
  hipMalloc((void**)&d_labels, MNIST_LABELS * train_dataset->batches[0].size * sizeof(FP));
  FP* d_xentropy_derivative;
  hipMalloc((void**)&d_xentropy_derivative, MNIST_LABELS * train_dataset->batches[0].size * sizeof(FP));

  uint32_t max_dim = 0;
  for(i=num_layers - 1; i>=2; i--) {
    if(layer_dims_arr[i - 1] > max_dim)
      max_dim = layer_dims_arr[i - 1];
  }
  backprop_args_t* backprop_args_arr = (backprop_args_t*)malloc(NUM_STREAMS * sizeof(backprop_args_t));
  for(i=0; i<NUM_STREAMS; i++) {
    hipMalloc((void**)&backprop_args_arr[i].d_curr_derivative, max_dim * sizeof(FP));
    hipMalloc((void**)&backprop_args_arr[i].d_next_derivative, (max_dim + 1) * sizeof(FP));
  }

  activation_deriv_args_t* actv_deriv_args_arr = (activation_deriv_args_t*)malloc(NUM_STREAMS * sizeof(activation_deriv_args_t));
  for(i=0; i<NUM_STREAMS; i++) {
    hipMalloc((void**)&actv_deriv_args_arr[i].d_softmax_jacobian, max_dim * max_dim * sizeof(FP));
    hipMalloc((void**)&actv_deriv_args_arr[i].d_layer_output_single, max_dim * sizeof(FP));
    hipMalloc((void**)&actv_deriv_args_arr[i].d_softmax_derivative, max_dim * sizeof(FP));
    hipMalloc((void**)&actv_deriv_args_arr[i].d_max_val, sizeof(FP));
    hipMalloc((void**)&actv_deriv_args_arr[i].d_sum, sizeof(FP));
    hipMalloc((void**)&actv_deriv_args_arr[i].d_relu_derivative, max_dim * sizeof(FP));
  }

  cat_xentropy_args_t xentropy_args;
  hipMalloc((void**)&xentropy_args.d_max_arr, train_dataset->batches[0].size * sizeof(FP));
  hipMalloc((void**)&xentropy_args.d_sum_arr, train_dataset->batches[0].size * sizeof(FP));
  hipHostMalloc((void**)&xentropy_args.xentropy_arr, train_dataset->batches[0].size * sizeof(FP), hipHostMallocDefault);
  hipMalloc((void**)&xentropy_args.d_xentropy_arr, train_dataset->batches[0].size * sizeof(FP));
  FP* max_arr = (FP*)malloc(train_dataset->batches[0].size * sizeof(FP));
  FP* sum_arr = (FP*)malloc(train_dataset->batches[0].size * sizeof(FP));
  for(i=0; i<train_dataset->batches[0].size; i++) {
    max_arr[i] = -INFINITY;
    sum_arr[i] = 0;
  }
  hipMemcpy(xentropy_args.d_max_arr, max_arr, train_dataset->batches[0].size * sizeof(FP), hipMemcpyHostToDevice);
  hipMemcpy(xentropy_args.d_sum_arr, sum_arr, train_dataset->batches[0].size * sizeof(FP), hipMemcpyHostToDevice);
  free(max_arr);
  free(sum_arr);

  FP** d_activation_deriv_arr = (FP**)malloc(NUM_STREAMS * sizeof(FP*));
  
  // MINI BATCH GRADIENT DESCENT
  omp_set_num_threads(NUM_STREAMS);
  #pragma omp parallel shared(train_dataset, layer_mtdt_arr, layer_dims_arr, num_layers, learning_rate, d_labels, xentropy_args, output_layer_activations, d_xentropy_derivative, backprop_params_arr, streams, backprop_args_arr, actv_deriv_args_arr, d_activation_deriv_arr) default(none)
  {
    int tid = omp_get_thread_num();
    mini_batch_gradient_descent(train_dataset, layer_mtdt_arr, layer_dims_arr, num_layers, learning_rate, d_labels,
                                &xentropy_args, output_layer_activations, d_xentropy_derivative, backprop_params_arr,
                                streams, backprop_args_arr, actv_deriv_args_arr, d_activation_deriv_arr, tid);
  }

  for(i=0; i<NUM_STREAMS; i++) {
    hipStreamDestroy(streams[i]);
  }
  free(output_layer_activations);
  free(backprop_params_arr);
  hipFree(d_labels);
  hipFree(d_xentropy_derivative);

  for(i=0; i<NUM_STREAMS; i++) {
    hipFree(backprop_args_arr[i].d_curr_derivative);
    hipFree(backprop_args_arr[i].d_next_derivative);
  }
  free(backprop_args_arr);

  for(i=0; i<NUM_STREAMS; i++) {
    hipFree(actv_deriv_args_arr[i].d_softmax_jacobian);
    hipFree(actv_deriv_args_arr[i].d_layer_output_single);
    hipFree(actv_deriv_args_arr[i].d_softmax_derivative);
    hipFree(actv_deriv_args_arr[i].d_max_val);
    hipFree(actv_deriv_args_arr[i].d_sum);
    hipFree(actv_deriv_args_arr[i].d_relu_derivative);
  }
  free(actv_deriv_args_arr);

  hipFree(xentropy_args.d_max_arr);
  hipFree(xentropy_args.d_sum_arr);
  hipHostFree(xentropy_args.xentropy_arr);
  hipFree(xentropy_args.d_xentropy_arr);

  free(d_activation_deriv_arr);
}

void test_mnist(mnist_dataset_t* test_dataset, layer_metadata_t* layer_mtdt_arr, uint32_t* layer_dims_arr, uint32_t num_layers) {
  dim3 Grid;
  dim3 Block(BLOCK_SZ, BLOCK_SZ); 
  int i, j;
  uint32_t actual_batch_size;
  FP* xentropy_arr;
  FP* output_layer_activations = (FP*)malloc(MNIST_LABELS * test_dataset->batches[0].size * sizeof(FP));
  FP batch_test_accuracy;
  FP overall_test_accuracy = 0;
  FP* d_labels;
  hipMalloc((void**)&d_labels, MNIST_LABELS * test_dataset->batches[0].size * sizeof(FP));

  cat_xentropy_args_t xentropy_args;
  hipMalloc((void**)&xentropy_args.d_max_arr, test_dataset->batches[0].size * sizeof(FP));
  hipMalloc((void**)&xentropy_args.d_sum_arr, test_dataset->batches[0].size * sizeof(FP));
  hipHostMalloc((void**)&xentropy_args.xentropy_arr, test_dataset->batches[0].size * sizeof(FP), hipHostMallocDefault);
  hipMalloc((void**)&xentropy_args.d_xentropy_arr, test_dataset->batches[0].size * sizeof(FP));
  FP* max_arr = (FP*)malloc(test_dataset->batches[0].size * sizeof(FP));
  FP* sum_arr = (FP*)malloc(test_dataset->batches[0].size * sizeof(FP));
  for(i=0; i<test_dataset->batches[0].size; i++) {
    max_arr[i] = -INFINITY;
    sum_arr[i] = 0;
  }
  hipMemcpy(xentropy_args.d_max_arr, max_arr, test_dataset->batches[0].size * sizeof(FP), hipMemcpyHostToDevice);
  hipMemcpy(xentropy_args.d_sum_arr, sum_arr, test_dataset->batches[0].size * sizeof(FP), hipMemcpyHostToDevice);
  free(max_arr);
  free(sum_arr);

  uint32_t* correct_cnt_arr = (uint32_t*)malloc(MNIST_LABELS * sizeof(uint32_t));
  for(i=0; i<MNIST_LABELS; i++)
    correct_cnt_arr[i] = 0;
 
  for(i=0; i<test_dataset->num_batches; i++) {
    actual_batch_size = test_dataset->batches[i].size; 
    hipMemcpy(layer_mtdt_arr[0].activations, test_dataset->batches[i].pixels, layer_dims_arr[0] * actual_batch_size * sizeof(FP), hipMemcpyHostToDevice);
    for(j=1; j<num_layers; j++) {
      Grid.x = (actual_batch_size + BLOCK_SZ - 1) / BLOCK_SZ;
      Grid.y = (layer_mtdt_arr[j].n + BLOCK_SZ - 1) / BLOCK_SZ;
      // Calculate outputs and ReLU activations for next layer
      blocked_gpu_matrixmult<<<Grid,Block>>>(layer_mtdt_arr[j].weights, layer_mtdt_arr[j-1].activations, layer_mtdt_arr[j].outputs, layer_mtdt_arr[j].activations, layer_mtdt_arr[j].n, actual_batch_size, layer_mtdt_arr[j].p, true);
    }

    // Apply softmax to the last layer and calculate categorical cross entropy loss
    hipMemcpy(d_labels, test_dataset->batches[i].labels, MNIST_LABELS * actual_batch_size * sizeof(FP), hipMemcpyHostToDevice);
    xentropy_arr = calc_categorical_xentropy(layer_mtdt_arr[num_layers - 1].outputs, layer_mtdt_arr[num_layers - 1].activations, actual_batch_size, d_labels, &xentropy_args);
    
    // Calculate test accuracy for batch
    hipMemcpy(output_layer_activations, layer_mtdt_arr[num_layers - 1].activations, MNIST_LABELS * actual_batch_size * sizeof(FP), hipMemcpyDeviceToHost);
    batch_test_accuracy = calc_accuracy(output_layer_activations, test_dataset->batches[i].labels, actual_batch_size, correct_cnt_arr);
    printf("Batch %d Test Accuracy: %.2f\n", i + 1, batch_test_accuracy);
    overall_test_accuracy += batch_test_accuracy;
    
    // PRINT START
    /*printf("\n\nBatch %d Cross Entropy\n\n", i);
    for(j=0; j<actual_batch_size; j++) {
      printf("%.2f ", xentropy_arr[j]);
    }
    printf("\n");*/
    // PRINT END
  }

  overall_test_accuracy /= (FP)test_dataset->num_batches;
  printf("Overall Test Accuracy: %.2f\n", overall_test_accuracy);
  printf("Correct Count:\n");
  for(i=0; i<MNIST_LABELS; i++) {
    printf("%d: %d\n", i, correct_cnt_arr[i]);
  }

  free(correct_cnt_arr);
  free(output_layer_activations);
  hipFree(d_labels);
  hipFree(xentropy_args.d_max_arr);
  hipFree(xentropy_args.d_sum_arr);
  hipHostFree(xentropy_args.xentropy_arr);
  hipFree(xentropy_args.d_xentropy_arr);
}

int main(int argc, char *argv[]) {
  int i; // loop counters

  int gpucount = 0; // Count of available GPUs
  int gpunum = 0; // Device number to use 
  
  uint32_t* layer_dims_arr;

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else {
     printf("Device count = %d\n",gpucount);
  }
   
  if (argc < 5) {
    printf("Usage: neural <epochs> <learning rate> <batch size> <layer dim 1> <layer dim 2> ... \n");
    exit (-1);
  }
  if (argc < 6) {
    printf("Must specify input and output layer dims\n");
    exit(-1);
  }
  if (argc < 7) {
    printf("Must specify at least one hidden layer\n");
    exit(-1);
  }

  if(BLOCK_SZ * BLOCK_SZ > 1024) {
    printf("Error, too many threads in block\n");
    exit(-1);
  }
  
  int epochs = atoi(argv[1]);
  FP learning_rate = (FP)atof(argv[2]);
  uint32_t batch_size = (uint32_t)atoi(argv[3]);
  uint32_t num_layers = (uint32_t)(argc - 4);
  
  // Training Dataset Initialization 
  uint8_t* train_images_raw = read_binary_file("../data/mnist/train-images-idx3-ubyte");
  uint8_t* train_labels_raw = read_binary_file("../data/mnist/train-labels-idx1-ubyte");
  // uint8_t* train_images_raw = read_binary_file("../data/cifar/train_images_cifar");
  // uint8_t* train_labels_raw = read_binary_file("../data/cifar/train_labels_cifar");
  
  mnist_image_file_header_t train_images_header = *((mnist_image_file_header_t*)train_images_raw);
  images_header_to_little_endian(&train_images_header); 
  if(train_images_header.magic_number != MNIST_IMAGE_MAGIC) {
    printf("Train image file magic (checksum) doesn't match.\n");
    exit(-1);
  }

  mnist_label_file_header_t train_labels_header = *((mnist_label_file_header_t*)train_labels_raw);
  labels_header_to_little_endian(&train_labels_header);
  if(train_labels_header.magic_number != MNIST_LABEL_MAGIC) {
    printf("Train label file magic (checksum) doesn't match.\n");
    exit(-1);
  }

  if(train_images_header.num_images != train_labels_header.num_labels) {
    printf("Number of images has to match number of labels!\n");
    exit(-1);
  }

  mnist_dataset_t train_dataset;
  init_dataset(&train_dataset, &train_images_header, train_images_raw, &train_labels_header, train_labels_raw, batch_size);
  free(train_images_raw);
  free(train_labels_raw);

  // Test Dataset Initialization
  uint8_t* test_images_raw = read_binary_file("../data/mnist/t10k-images-idx3-ubyte");
  uint8_t* test_labels_raw = read_binary_file("../data/mnist/t10k-labels-idx1-ubyte");
  // uint8_t* test_images_raw = read_binary_file("../data/cifar/test_images_cifar");
  // uint8_t* test_labels_raw = read_binary_file("../data/cifar/test_labels_cifar");
  
  mnist_image_file_header_t test_images_header = *((mnist_image_file_header_t*)test_images_raw);
  images_header_to_little_endian(&test_images_header); 
  if(test_images_header.magic_number != MNIST_IMAGE_MAGIC) {
    printf("Train image file magic (checksum) doesn't match.\n");
    exit(-1);
  }

  mnist_label_file_header_t test_labels_header = *((mnist_label_file_header_t*)test_labels_raw);
  labels_header_to_little_endian(&test_labels_header);
  if(test_labels_header.magic_number != MNIST_LABEL_MAGIC) {
    printf("Train label file magic (checksum) doesn't match.\n");
    exit(-1);
  }

  if(test_images_header.num_images != test_labels_header.num_labels) {
    printf("Number of images has to match number of labels!\n");
    exit(-1);
  }

  mnist_dataset_t test_dataset;
  init_dataset(&test_dataset, &test_images_header, test_images_raw, &test_labels_header, test_labels_raw, batch_size);
  free(test_images_raw);
  free(test_labels_raw);
  
  layer_dims_arr = (uint32_t*)malloc(num_layers * sizeof(uint32_t));
  for(i=0; i<num_layers; i++) {
    layer_dims_arr[i] = (uint32_t)atoi(argv[i + 4]);
  }
  if(layer_dims_arr[0] != train_images_header.num_rows * train_images_header.num_cols) {
    printf("Number of nodes in first layer must match pixel count of each image in dataset.\n");
    exit(-1);
  }
  
  hipSetDevice(gpunum);
  printf("Using device %d\n", gpunum);
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);

  layer_metadata_t* layer_mtdt_arr = init_layer_metadata_arr(layer_dims_arr, num_layers, batch_size);
  
  // TRAIN
  for(i=0; i<epochs; i++) {
    printf("\n\n******EPOCH %d******\n\n", i+1);
    train_mnist(&train_dataset, layer_mtdt_arr, layer_dims_arr, num_layers, learning_rate);
  }

  // TEST
  printf("\n\n******TEST******\n\n");
  test_mnist(&test_dataset, layer_mtdt_arr, layer_dims_arr, num_layers);
  
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop);  

  printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time
  
  free(layer_dims_arr);
  free_layer_metadata_arr(layer_mtdt_arr, num_layers);
  free_dataset(&train_dataset);
  free_dataset(&test_dataset);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
